#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "util.h"

__global__ void printThreadIndex(float *A,const int nx,const int ny)
{
  int ix=threadIdx.x+blockIdx.x*blockDim.x;//blockDim.x和y为线程的维度4x2
  int iy=threadIdx.y+blockIdx.y*blockDim.y;
  unsigned int idx=iy*nx+ix;
  printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d)"
          "global index %2d boockdim.x:%d blockdim.y:%d\n",threadIdx.x,threadIdx.y,
          blockIdx.x,blockIdx.y,ix,iy,idx,blockDim.x,blockDim.y);
}
int main(int argc,char** argv)
{
  initDevice(0);
  int nx=8,ny=6;
  int nxy=nx*ny;
  int nBytes=nxy*sizeof(float);

  //Malloc
  float* A_host=(float*)malloc(nBytes);
  initialData(A_host,nxy);
  printMatrix(A_host,nx,ny);

  //hipMalloc
  float *A_dev=NULL;
  CHECK(hipMalloc((void**)&A_dev,nBytes));

  hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice);

  dim3 block(4,2);//8个线程
  dim3 grid((nx-1)/block.x+1,(ny-1)/block.y+1);//
  printThreadIndex<<<grid,block>>>(A_dev,nx,ny);
  CHECK(hipDeviceSynchronize());
  hipFree(A_dev);
  free(A_host);

  hipDeviceReset();
  return 0;
}