#include "hip/hip_runtime.h"
/*
* https://github.com/Tony-Tan/CUDA_Freshman
* 3_sum_arrays
*/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "util.h"


void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
}
__global__ void sumArraysGPU(float*a,float*b,float*res)
{
  int i=threadIdx.x;
  res[i]=a[i]+b[i];
}
int main(int argc,char **argv)
{
  int dev = 2;
  hipSetDevice(dev);

  int nElem=32;
  double iStart,iElaps;
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *a_h=(float*)malloc(nByte);
  float *b_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte);
  float *res_from_gpu_h=(float*)malloc(nByte);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_d,*b_d,*res_d;
  CHECK(hipMalloc((float**)&a_d,nByte));
  CHECK(hipMalloc((float**)&b_d,nByte));
  CHECK(hipMalloc((float**)&res_d,nByte));

  initialData(a_h,nElem);
  initialData(b_h,nElem);

  CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

  dim3 block(nElem);
  dim3 grid(nElem/block.x);
  /*
    大概分析下核函数启动到结束的过程：
    主机线程启动核函数
    核函数启动成功
    控制返回主机线程
    核函数执行完成
    主机同步函数侦测到核函数执行完
    要测试的是2~4的时间，但是用CPU计时方法，只能测试1~5的时间
  */
  iStart=cpuSecond();
  sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d);
  hipDeviceSynchronize();
  iElaps=cpuSecond()-iStart;
  printf("Execution configuration<<<%d,%d>>>\n",block.x,grid.x);
  printf("Execution time:%f\n",iElaps);
  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  sumArrays(a_h,b_h,res_h,nElem);

  checkResult(res_h,res_from_gpu_h,nElem);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(res_d);

  free(a_h);
  free(b_h);
  free(res_h);
  free(res_from_gpu_h);

  return 0;
}